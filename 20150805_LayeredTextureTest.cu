#include "hip/hip_runtime.h"
/*
This program is used test how the texture memory is arrange on the global memeory
so as to find the fastest access pattern.

*/

#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <string>
#include <fstream>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>


#define PIC_WIDTH 1024
#define PIC_HEIGHT 1024
static const int picSize = PIC_WIDTH * PIC_HEIGHT;
#define picLayerNum 64

struct texObjtStrut { hipTextureObject_t texAry[picLayerNum]; };
/*pattern 1 */
__global__ void readTextureLayerByLayer(float *output, hipTextureObject_t LayerTex){

	float ftemp = 0.0f;
#pragma unroll
	for (int layer = 0; layer < picLayerNum; layer++){
		for (int row = 0; row < PIC_HEIGHT; row++) {
			for (int col = 0; col < PIC_WIDTH; col++) {
				ftemp += tex2DLayered<float>(LayerTex, row, col, layer);
			}
		}
	}
	*output = ftemp;
}
/*pattern 2*/
__global__ void readTextureDotByDot(float *output, hipTextureObject_t LayerTex){

	float ftemp = 0.0f;

	for (int row = 0; row < PIC_HEIGHT; row++) {
		for (int col = 0; col < PIC_WIDTH; col++) {
#pragma unroll
			for (int layer = 0; layer < picLayerNum; layer++){
				ftemp += tex2DLayered<float>(LayerTex, row, col, layer);
			}
		}
	}
	*output = ftemp;
}
/*pattern 3*/
__global__ void readTextureObjtByObjt(float *output, texObjtStrut texObjtSet){

	float ftemp = 0.0f;
#pragma unroll
	for (int objt = 0; objt < picLayerNum; objt++){
		for (int row = 0; row < PIC_HEIGHT; row++) {
			for (int col = 0; col < PIC_WIDTH; col++) {
				ftemp += tex2D<float>(texObjtSet.texAry[objt], row, col);
			}
		}
	}
	*output = ftemp;
}

/*pattern4 put different pic in diferent tex obj and
access the tex by the loop sequence of row->col->tex*/
__global__ void pattern4(float *output, texObjtStrut texObjtSet){

	float ftemp = 0.0f;

	for (int row = 0; row < PIC_HEIGHT; row++) {
		for (int col = 0; col < PIC_WIDTH; col++) {
#pragma unroll
			for (int objt = 0; objt < picLayerNum; objt++){
				ftemp += tex2D<float>(texObjtSet.texAry[objt], row, col);
			}
		}
	}
	*output = ftemp;
}
/*pattern 5 is all the same with pattern 3 except from the unroll setup*/
__global__ void pattern5(float *output, texObjtStrut texObjtSet){

	float ftemp = 0.0f;

	for (int objt = 0; objt < picLayerNum; objt++){
		for (int row = 0; row < PIC_HEIGHT; row++) {
#pragma unroll 100
			for (int col = 0; col < PIC_WIDTH; col++) {
				ftemp += tex2D<float>(texObjtSet.texAry[objt], row, col);
			}
		}
	}
	*output = ftemp;
}

int main()
{
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("kernelExecTimeoutEnabled = %d\n",prop.kernelExecTimeoutEnabled);
	srand(2015);

	float *pictureSET;
	checkCudaErrors(hipHostAlloc((void**)&pictureSET, sizeof(float) * picLayerNum * PIC_WIDTH * PIC_HEIGHT, hipHostMallocDefault));
	for (int i = 0; i < picLayerNum * PIC_WIDTH * PIC_HEIGHT; i++){
		pictureSET[i] = rand() / (float)RAND_MAX;
	}

	float hostmem = 0.0f;


	hipArray_t tex_buf;
	hipChannelFormatDesc desc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipExtent extent;
	extent.width = PIC_WIDTH;
	extent.height = PIC_HEIGHT;
	extent.depth = picLayerNum;
	checkCudaErrors(hipMalloc3DArray(&tex_buf, &desc, extent, hipArrayLayered));

	// generate texture object for reading
	hipTextureObject_t         texInput;
	hipResourceDesc            texRes;
	memset(&texRes, 0, sizeof(hipResourceDesc));

	texRes.resType = hipResourceTypeArray;
	texRes.res.array.array = tex_buf;

	hipTextureDesc             texDescr;
	memset(&texDescr, 0, sizeof(hipTextureDesc));

	texDescr.normalizedCoords = 0;		//Indicates whether texture reads are normalized or not
	texDescr.filterMode = hipFilterModeLinear;

	texDescr.addressMode[0] = hipAddressModeClamp;
	texDescr.addressMode[1] = hipAddressModeClamp;
	texDescr.addressMode[2] = hipAddressModeClamp;
	texDescr.readMode = hipReadModeElementType;  /**< Read texture as specified element type */
	checkCudaErrors(hipCreateTextureObject(&texInput, &texRes, &texDescr, NULL));

	/*---------------- for copy data --------------------- */
	hipMemcpy3DParms myparms = { 0 };
	myparms.srcPos = make_hipPos(0, 0, 0);
	myparms.dstPos = make_hipPos(0, 0, 0);
	myparms.srcPtr = make_hipPitchedPtr(pictureSET, PIC_WIDTH * sizeof(float), PIC_WIDTH, PIC_HEIGHT);
	myparms.dstArray = tex_buf;
	myparms.extent = make_hipExtent(PIC_WIDTH, PIC_HEIGHT, picLayerNum);
	myparms.kind = hipMemcpyHostToDevice;

	checkCudaErrors(hipMemcpy3D(&myparms));


	float *deviceMem;
	checkCudaErrors(hipMalloc((void**)&deviceMem, sizeof(float)));

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float elapsedTime;

	dim3 grid_tex(1, 1);
	dim3 thread_tex(1, 1);

	hipEventRecord(start, 0);
	readTextureLayerByLayer << < grid_tex, thread_tex >> > (deviceMem, texInput);
	getLastCudaError("CUDA kernel failed\n");
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("time used layerbylayer (pattern 1) = %0.0f\n", elapsedTime);
	checkCudaErrors(hipMemcpy(&hostmem, deviceMem, sizeof(float), hipMemcpyDeviceToHost));
	printf("result = %f\n", hostmem);
	memset(&hostmem, 0, sizeof(hostmem));

	hipEventRecord(start, 0);
	readTextureDotByDot << < grid_tex, thread_tex >> > (deviceMem, texInput);
	getLastCudaError("CUDA kernel failed\n");
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("time used dotbydot (pattern 2) = %0.0f\n", elapsedTime);
	checkCudaErrors(hipMemcpy(&hostmem, deviceMem, sizeof(float), hipMemcpyDeviceToHost));
	printf("result = %f\n", hostmem);
	memset(&hostmem, 0, sizeof(hostmem));
	
	checkCudaErrors(hipFreeArray(tex_buf));
	checkCudaErrors(hipDestroyTextureObject(texInput));

	float* tex_data[picLayerNum];
	size_t pitch[picLayerNum];

	hipResourceDesc resDesc;
	hipTextureDesc texDesc;
	texObjtStrut texContainer;

	for (int i = 0; i < picLayerNum; i++){
		hipMallocPitch(&tex_data[i], &pitch[i], sizeof(float)*PIC_WIDTH, PIC_HEIGHT);
		memset(&resDesc, 0, sizeof(resDesc));
		resDesc.resType = hipResourceTypePitch2D;
		resDesc.res.pitch2D.devPtr = tex_data[i];
		resDesc.res.pitch2D.pitchInBytes = pitch[i];
		resDesc.res.pitch2D.width = PIC_WIDTH;
		resDesc.res.pitch2D.height = PIC_HEIGHT;
		resDesc.res.pitch2D.desc = hipCreateChannelDesc<float>();
		memset(&texDesc, 0, sizeof(texDesc));
		texDesc.readMode = hipReadModeElementType;
		texDesc.addressMode[0] = hipAddressModeClamp;// X axis
		texDesc.addressMode[1] = hipAddressModeClamp;// Y axis
		texDesc.filterMode = hipFilterModeLinear;
		hipCreateTextureObject(&texContainer.texAry[i], &resDesc, &texDesc, NULL);
		checkCudaErrors(hipMemcpy2D(tex_data[i], pitch[i], &pictureSET[i],
			sizeof(float) * PIC_WIDTH, sizeof(float) * PIC_WIDTH,
			PIC_HEIGHT, hipMemcpyHostToDevice));
	}



	
	
	hipEventRecord(start, 0);
	readTextureObjtByObjt << < grid_tex, thread_tex >> > (deviceMem, texContainer);	
	getLastCudaError("CUDA kernel failed\n");
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("time used objtbyobjt (pattern 3) = %0.0f\n", elapsedTime);
	checkCudaErrors(hipMemcpy(&hostmem, deviceMem, sizeof(float), hipMemcpyDeviceToHost));
	printf("result = %f\n", hostmem);
	memset(&hostmem, 0, sizeof(hostmem));

	hipEventRecord(start, 0);
	pattern4 << < grid_tex, thread_tex >> > (deviceMem, texContainer);
	getLastCudaError("CUDA kernel failed\n");
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("time used pattern4  = %0.0f\n", elapsedTime);
	(hipMemcpy(&hostmem, deviceMem, sizeof(float), hipMemcpyDeviceToHost));
	printf("result = %f\n", hostmem);

	hipEventRecord(start, 0);
	pattern5 << < grid_tex, thread_tex >> > (deviceMem, texContainer);
	getLastCudaError("CUDA kernel failed\n");
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, stop));
	printf("time used pattern 5  = %0.0f\n", elapsedTime);
	(hipMemcpy(&hostmem, deviceMem, sizeof(float), hipMemcpyDeviceToHost));
	printf("result = %f\n", hostmem);


	for (int i = 0; i < picLayerNum; i++){
		checkCudaErrors(hipFree(tex_data[i]));//
		checkCudaErrors(hipDestroyTextureObject(texContainer.texAry[i]));
	}

	checkCudaErrors(hipHostFree(pictureSET));
	hipEventDestroy(start);	hipEventDestroy(stop);
	hipDeviceReset();
	return 0;
}
